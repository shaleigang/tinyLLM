#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include <algorithm>
#include <cassert>
#include <cfloat>
#include <cmath>
#include <cstring>
#include <numeric>

#include "function.h"

namespace tllm {
namespace F {
detail::MatMulExp mat_mul;
detail::LayerNorm layer_norm(0.00001);
detail::Softmax softmax;
detail::Log log;
detail::NLLLoss nll_loss;
detail::Emb emb;

Tensor cross_entropy(Tensor& x1, Tensor& x2) {
  Tensor x1_softmax = softmax(x1);
  Tensor x1_softmax_log = log(x1_softmax);
  return nll_loss(x1_softmax_log, x2);
}

__global__ void causal_mask_fill_kernel(float* att, index_t T, index_t dsize);
void causal_mask_fill(Tensor& att) {
  // att (B, nh, T, T)
  auto shape = att.shape();
  assert(shape.size() == 4);
  assert(shape[2] == shape[3]);

  index_t B = shape[0];
  index_t nh = shape[1];
  index_t T = shape[2];

  if (att.device() == "cpu") {
    for (index_t b = 0; b < B; ++b) {
      for (index_t h = 0; h < nh; ++h) {
        for (index_t i = 0; i < T; ++i) {
          for (index_t j = i + 1; j < T; ++j) {
            att[{b, h, i, j}] = FLT_MIN;
          }
        }
      }
    }
  } else {
    dim3 grid_size((T + 25) / 26, (T + 25) / 26, B * nh);
    dim3 block_size(26, 26);
    causal_mask_fill_kernel<<<grid_size, block_size>>>(att.data(), T, att.dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

__global__ void causal_mask_fill_kernel(float* att, index_t T, index_t dsize) {
  // printf("%d %d %d\n", gridDim.x, gridDim.y, gridDim.z);
  const int head_idx = blockIdx.z * T * T;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int x = blockIdx.x * blockDim.y + threadIdx.x;
  if (y < T && x < T && x > y) {
      att[head_idx + y * T + x] = FLT_MIN;
  }
}

}  // namespace F
}  // namespace tllm

using namespace tllm::detail;

using std::placeholders::_1;
using std::placeholders::_2;
using std::placeholders::_3;

__global__ void matmul_kernel(float* x, float* x1, float* x2, index_t z_,
                              index_t y_, index_t x_, index_t d_,
                              index_t dsize2);
__global__ void matmul_backward_kernel(float* x, float* x1, float* x2,
                                       index_t z_, index_t y_, index_t x_,
                                       index_t d_, index_t dsize,
                                       index_t dsize1, index_t dsize2);
__global__ void layer_norm_kernel(float* x1, float* x, int N, index_t dim,
                                  index_t dsize, float ep);
__global__ void layer_norm_backward_kernel(float* x1_grad, float* x1_data,
                                           float* x_grad, float* x_data, int N,
                                           index_t dim, index_t dsize,
                                           float ep);
__global__ void drouput_kernel(float* x1, float* x, float prob, index_t dsize,
                               unsigned long long seed);
__global__ void dropout_backward_kernel(float* x1_grad, float* x_grad,
                                        float* x_data, float prob,
                                        index_t dsize);
__global__ void gelu_kernel(float* x1, float* x, index_t dsize);
__global__ void gelu_backward_kernel(float* x1_grad, float* x1_data,
                                     float* x_grad, index_t dsize);
__global__ void softmax_kernel(float* x1, float* x, int N, index_t dim,
                               index_t dsize);
__global__ void softmax_kernel(float* x1, float* x, int N, index_t dim,
                               index_t dsize);
__global__ void softmax_backward_kernel(float* x1_grad, float* x1_data,
                                        float* x_grad, float* x_data, int N,
                                        index_t dim, index_t dsize);
__global__ void log_kernel(float* x1, float* x, index_t dsize);
__global__ void log_backward_kernel(float* x1_grad, float* x1_data,
                                    float* x_grad, index_t dsize);
__global__ void nllloss_kernel(float* x1, float* x2, float* x, index_t len,
                               index_t dsize);
__global__ void nllloss_backward_kernel(float* x1_grad, float* x1_data,
                                        float* x2_data, float* x_grad,
                                        index_t len, index_t dsize);
__global__ void emb_backward_kernel(float* idx, float* emb_grad, float* x_grad,
                                    int N, index_t dim, index_t dsize);

Tensor UnaryFunc::operator()(Tensor& x1) { return forward(x1); }

Tensor BinaryFunc::operator()(Tensor& x1, Tensor& x2) {
  return forward(x1, x2);
}

Tensor MatMulExp::generate_ret_tensor(Tensor& x1, Tensor& x2) {
  auto shape1 = x1.shape();
  auto shape2 = x2.shape();
  if (shape1[shape1.size() - 1] != shape2[shape2.size() - 2]) {
    std::cout << "MatMul shape not match." << std::endl;
    assert(false);
  }
  shape1[shape1.size() - 1] = shape2[shape2.size() - 1];
  return Tensor(shape1, x1.device());
}

void MatMulExp::prepare_forward(Tensor& x1, Tensor& x2, Tensor& x) {
  // 1. check
  if (x1.device() != x2.device()) {
    std::cout << "TensorImplPtr must in same device." << std::endl;
    assert(false);
  }

  if (x1.ndim() < x2.ndim()) {
    std::cout << "LHS ndim smaller than RHS ndim. TensorImplPtr not in same "
                 "shape and can not broadcast."
              << std::endl;
    assert(false);
  }
  index_t l = x2.ndim();
  for (int i = 3; i <= l; ++i) {
    if (x1.shape()[x1.ndim() - i] != x2.shape()[x2.ndim() - i]) {
      std::cout << x1.shape()[x1.ndim() - i] << " " << x2.shape()[x2.ndim() - i]
                << std::endl;
      std::cout << "TensorImplPtr not in same shape and can not broadcast."
                << std::endl;
      assert(false);
    }
  }

  // 2. add node to return Tensor
  GraphNodePtr node = std::make_shared<BinaryGraphNode>(x1.get(), x2.get());
  node->setGradFnL(std::bind(&MatMulExp::lhs_grad_fn, this, _1, _2, _3));
  node->setGradFnR(std::bind(&MatMulExp::rhs_grad_fn, this, _1, _2, _3));
  x.setNode(node);

  x2.transpose(x2.ndim() - 2, x2.ndim() - 1);

  return;
}

void MatMulExp::forward_process(Tensor& x1, Tensor& x2, Tensor& x) {
  auto shape1 = x1.shape();
  auto shape2 = x2.shape();
  auto shape = x.shape();
  index_t dim = 1;
  for (int i = 0; i < shape.size() - 2; ++i) {
    dim *= shape[i];
  }
  index_t dim2 = 1;
  for (int i = 0; i < shape2.size() - 2; ++i) {
    dim2 *= shape2[i];
  }
  x1.view({dim, shape1[shape1.size() - 2], shape1[shape1.size() - 1]});
  x2.view({dim2, shape2[shape2.size() - 2], shape2[shape2.size() - 1]});
  x.view({dim, shape[shape.size() - 2], shape[shape.size() - 1]});
  if (x1.device() == "cpu") {
    for (index_t m = 0; m < dim; ++m) {
      for (index_t i = 0; i < shape[shape.size() - 2]; ++i) {
        for (index_t j = 0; j < shape[shape.size() - 1]; ++j) {
          x[{m, i, j}] = 0;
          for (index_t p = 0; p < shape1[shape1.size() - 1]; ++p) {
            x[{m, i, j}] += (x1[{m, i, p}] * x2[{m % dim2, j, p}]);
          }
        }
      }
    }
  } else {
    const dim3 grid_size((shape[shape.size() - 1] + 15) / 16,
                         (shape[shape.size() - 2] + 15) / 16, dim);
    const dim3 block_size(16, 16);
    // std::cout << dim << " " << shape[shape.size() - 2] << " " <<
    // shape[shape.size() - 1] << " " << shape1[shape1.size() - 1] << std::endl;
    matmul_kernel<<<grid_size, block_size>>>(
        x.data(), x1.data(), x2.data(), dim, shape[shape.size() - 2],
        shape[shape.size() - 1], shape1[shape1.size() - 1], x2.dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
  x1.view(shape1);
  x2.view(shape2);
  x.view(shape);
  x2.transpose(x2.ndim() - 2, x2.ndim() - 1);
  return;
}

__global__ void matmul_kernel(float* x, float* x1, float* x2, index_t z_,
                              index_t y_, index_t x_, index_t d_,
                              index_t dsize2) {
  const int id_z = blockIdx.z;
  const int id_y = blockIdx.y * blockDim.y + threadIdx.y;
  const int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int x_offset = id_z * y_ * x_ + id_y * x_ + id_x;
  const int x1_offset = id_z * y_ * d_ + id_y * d_;
  const int x2_offset = id_z * x_ * d_ + id_x * d_;
  if (id_z < z_ && id_y < y_ && id_x < x_) {
    x[x_offset] = 0;
    // printf("%d %d %d %d %d\n", x_offset, id_z, id_y, id_x, gridDim.y);
    for (int i = 0; i < d_; ++i) {
      x[x_offset] += x1[x1_offset + i] * x2[(x2_offset + i) % dsize2];
    }
  }
}

void MatMulExp::lhs_grad_fn(TensorImplPtr x1, TensorImplPtr x2,
                            TensorImplPtr x) {
  x1->contiguous();
  x2->contiguous();
  x->contiguous();
  auto shape1 = x1->shape();
  auto shape2 = x2->shape();
  auto shape = x->shape();

  index_t dim = 1;
  for (int i = 0; i < shape.size() - 2; ++i) {
    dim *= shape[i];
  }
  index_t dim2 = 1;
  for (int i = 0; i < shape2.size() - 2; ++i) {
    dim2 *= shape2[i];
  }
  x1->view({dim, shape1[shape1.size() - 2], shape1[shape1.size() - 1]});
  x2->view({dim2, shape2[shape2.size() - 2], shape2[shape2.size() - 1]});
  x->view({dim, shape[shape.size() - 2], shape[shape.size() - 1]});

  if (x1->device() == "cpu") {
    for (index_t m = 0; m < dim; ++m) {
      for (index_t i = 0; i < shape[shape.size() - 2]; ++i) {
        for (index_t p = 0; p < shape1[shape1.size() - 1]; ++p) {
          for (index_t j = 0; j < shape[shape.size() - 1]; ++j) {
            x1->grad_[x1->get_offset({m, i, p})] +=
                (x->grad_[x->get_offset({m, i, j})] *
                 x2->data_[x2->get_offset({m % dim2, p, j})]);
          }
        }
      }
    }
  } else {
    const dim3 grid_size((shape1[shape1.size() - 1] + 15) / 16,
                         (shape1[shape1.size() - 2] + 15) / 16, dim);
    const dim3 block_size(16, 16);
    matmul_backward_kernel<<<grid_size, block_size>>>(
        x1->grad_, x->grad_, x2->data_, dim, shape1[shape1.size() - 2],
        shape1[shape1.size() - 1], shape2[shape2.size() - 1], x1->dsize(),
        x->dsize(), x2->dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
  x1->view(shape1);
  x2->view(shape2);
  x->view(shape);

  return;
}

__global__ void matmul_backward_kernel(float* x, float* x1, float* x2,
                                       index_t z_, index_t y_, index_t x_,
                                       index_t d_, index_t dsize,
                                       index_t dsize1, index_t dsize2) {
  const int id_z = blockIdx.z;
  const int id_y = blockIdx.y * blockDim.y + threadIdx.y;
  const int id_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int x_offset = id_z * y_ * x_ + id_y * x_ + id_x;
  const int x1_offset = id_z * y_ * d_ + id_y * d_;
  const int x2_offset = id_z * x_ * d_ + id_x * d_;
  if (id_z < z_ && id_y < y_ && id_x < x_) {
    for (int i = 0; i < d_; ++i) {
      atomicAdd(x + x_offset % dsize,
                x1[(x1_offset + i) % dsize1] * x2[(x2_offset + i) % dsize2]);
      // printf("%.2f %.2f %.2f %d\n", x[x_offset % dsize], x1[(x1_offset + i) %
      // dsize1], x2[(x2_offset + i) % dsize2], x_offset);
    }
  }
}

void MatMulExp::rhs_grad_fn(TensorImplPtr x1, TensorImplPtr x2,
                            TensorImplPtr x) {
  auto shape1 = x1->shape();
  auto shape2 = x2->shape();
  auto shape = x->shape();
  x1->transpose(x1->ndim() - 2, x1->ndim() - 1);
  x->transpose(x->ndim() - 2, x->ndim() - 1);
  x1->contiguous();
  x2->contiguous();
  x->contiguous();
  index_t dim = 1;
  for (int i = 0; i < shape.size() - 2; ++i) {
    dim *= shape[i];
  }
  index_t dim2 = 1;
  for (int i = 0; i < shape2.size() - 2; ++i) {
    dim2 *= shape2[i];
  }

  x1->view({dim, shape1[shape1.size() - 1], shape1[shape1.size() - 2]});
  x2->view({dim2, shape2[shape2.size() - 2], shape2[shape2.size() - 1]});
  x->view({dim, shape[shape.size() - 1], shape[shape.size() - 2]});
  if (x1->device() == "cpu") {
    for (index_t m = 0; m < dim; ++m) {
      for (index_t i = 0; i < shape[shape.size() - 2]; ++i) {
        for (index_t j = 0; j < shape[shape.size() - 1]; ++j) {
          for (index_t p = 0; p < shape1[shape1.size() - 1]; ++p) {
            x2->grad_[x2->get_offset({m % dim2, p, j})] +=
                (x1->data_[x1->get_offset({m, p, i})] *
                 x->grad_[x->get_offset({m, j, i})]);
          }
        }
      }
    }
  } else {
    const dim3 grid_size((shape2[shape2.size() - 1] + 15) / 16,
                         (shape2[shape2.size() - 2] + 15) / 16, dim);
    const dim3 block_size(16, 16);
    matmul_backward_kernel<<<grid_size, block_size>>>(
        x2->grad_, x1->data_, x->grad_, dim, shape2[shape2.size() - 2],
        shape2[shape2.size() - 1], shape1[shape1.size() - 2], x2->dsize(),
        x1->dsize(), x->dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }

  x1->transpose(x1->ndim() - 2, x1->ndim() - 1);
  x->transpose(x->ndim() - 2, x->ndim() - 1);

  x1->contiguous();
  x->contiguous();

  x1->view(shape1);
  x2->view(shape2);
  x->view(shape);

  // x2->decreaseRef();
  return;
}

void LayerNorm::forward_process(Tensor& x1, Tensor& x) {
  if (x1.device() == "cpu") {
    std::memcpy(x.data(), x1.data(), sizeof(float) * x1.dsize());
    index_t dim = x1.shape()[x1.shape().size() - 1];
    for (int b = 0; b < x1.dsize() / dim; ++b) {
      index_t offset = b * dim;
      float mean =
          std::accumulate(x1.data() + offset, x1.data() + offset + dim, 0.0) /
          dim;
      float accum = 0.0;
      std::for_each(x1.data() + offset, x1.data() + offset + dim,
                    [&](const float d) { accum += (d - mean) * (d - mean); });

      float stdev = sqrt(accum / dim + ep_);
      std::for_each(x.data() + offset, x.data() + offset + dim, [&](float& d) {
        d = d - mean;
        d = d / stdev;
      });
    }
  } else {
    auto shape = x.shape();
    // hipMemcpy(x.data(), x1.data(), sizeof(float) * x1.dsize(),
    // hipMemcpyDeviceToDevice);
    const dim3 grid_size(
        1, shape[shape.size() - 2],
        x.dsize() / (shape[shape.size() - 2] * shape[shape.size() - 1]));
    const int block_size = min(
        512, (shape[shape.size() - 1] % 2 == 0 ? shape[shape.size() - 1]
                                               : shape[shape.size() - 1] + 1));
    layer_norm_kernel<<<grid_size, block_size,
                        sizeof(float) * shape[shape.size() - 1]>>>(
        x1.data(), x.data(),
        (shape[shape.size() - 1] + block_size - 1) / block_size,
        shape[shape.size() - 1], x.dsize(), ep_);
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

__global__ void layer_norm_kernel(float* x1, float* x, int N, index_t dim,
                                  index_t dsize, float ep) {
  const index_t z = blockIdx.z;
  const index_t y = blockIdx.y;
  const index_t thread_idx = threadIdx.x;

  const index_t head_offset = z * gridDim.y * dim + y * dim;

  extern __shared__ float data[];
  for (int i = 0; i < N; ++i) {
    index_t offset = thread_idx * N + i;
    if (offset < dim) {
      if ((head_offset + offset) < dsize) {
        data[offset] = x1[head_offset + offset];
      } else {
        data[offset] = 0;
      }
    } else {
      data[offset] = 0;
    }
  }
  __syncthreads();
  for (int gap = (blockDim.x * N) >> 1; gap > 0; gap >>= 1) {
    for (int i = 0; i < N; ++i) {
      int offset = thread_idx * N + i;
      if (offset < gap) {
        data[offset] += data[offset + gap];
      }
    }
    __syncthreads();
  }
  __syncthreads();
  float mean = data[0] / dim;

  for (int i = 0; i < N; ++i) {
    int offset = thread_idx * N + i;
    if (offset < dim && head_offset + offset < dsize) {
      data[offset] =
          (x1[head_offset + offset] - mean) * (x1[head_offset + offset] - mean);
    } else {
      data[offset] = 0;
    }
  }
  __syncthreads();
  for (int gap = blockDim.x * N >> 1; gap > 0; gap >>= 1) {
    for (int i = 0; i < N; ++i) {
      int offset = thread_idx * N + i;
      if (offset < gap) {
        data[offset] += data[offset + gap];
      }
    }
    __syncthreads();
  }
  __syncthreads();
  float accum = data[0];
  float stdev = sqrt(accum / dim + ep);

  for (int i = 0; i < N; ++i) {
    int offset = thread_idx * N + i;
    if (offset < dim && head_offset + offset < dsize) {
      x[head_offset + offset] = (x1[head_offset + offset] - mean) / stdev;
    }
  }
}

void LayerNorm::grad_fn(TensorImplPtr x1, TensorImplPtr x) {
  if (x1->device() == "cpu") {
    index_t dim = x1->shape()[x1->shape().size() - 1];
    for (int b = 0; b < x1->dsize() / dim; ++b) {
      index_t offset = b * dim;
      float mean =
          std::accumulate(x1->data_ + offset, x1->data_ + offset + dim, 0.0) /
          dim;
      float accum = 0.0;
      std::for_each(x1->data_ + offset, x1->data_ + offset + dim,
                    [&](const float d) { accum += (d - mean) * (d - mean); });

      float stdev = sqrt(accum / dim + ep_);

      float a1 = -(1 / (dim * stdev));
      // float a2 = stdev * (accum / dim + ep_);

      float grad_accum =
          std::accumulate(x->grad_ + offset, x->grad_ + offset + dim, 0.0) /
          (dim * stdev);
      grad_accum *= a1;

      float grad_data_accum = 0.0;
      for (int i = 0; i < dim; ++i) {
        grad_data_accum += x->grad_[offset + i] * x->data_[offset + i];
      }
      grad_data_accum *= a1;

      for (int i = 0; i < dim; ++i) {
        x1->grad_[offset + i] += (x->grad_[offset + i] / stdev + grad_accum +
                                  x->data_[offset + i] * grad_data_accum);
      }
    }
  } else {
    auto shape = x->shape();
    const dim3 grid_size(
        1, shape[shape.size() - 2],
        x->dsize() / (shape[shape.size() - 2] * shape[shape.size() - 1]));
    const int block_size = min(
        512, (shape[shape.size() - 1] % 2 == 0 ? shape[shape.size() - 1]
                                               : shape[shape.size() - 1] + 1));
    layer_norm_backward_kernel<<<grid_size, block_size,
                                 sizeof(float) * shape[shape.size() - 1]>>>(
        x1->grad_, x1->data_, x->grad_, x->data_,
        (shape[shape.size() - 1] + block_size - 1) / block_size,
        shape[shape.size() - 1], x->dsize(), ep_);
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
  // x->to("cpu");
  // x1->to("cpu");
  // for (int i = 0; i < x->dsize(); ++i) {
  //   if (std::isnan(x->grad_[i])) {
  //     std::cout << "before ln grad has nan." << std::endl;
  //     exit(0);
  //   }
  // }
  // for (int i = 0; i < x1->dsize(); ++i) {
  //   if (std::isnan(x1->grad_[i])) {
  //     std::cout << "after ln grad has nan." << std::endl;
  //     exit(0);
  //   }
  // }
  // x->to("cuda");
  // x1->to("cuda");
}

__global__ void layer_norm_backward_kernel(float* x1_grad, float* x1_data,
                                           float* x_grad, float* x_data, int N,
                                           index_t dim, index_t dsize,
                                           float ep) {
  const int z = blockIdx.z;
  const int y = blockIdx.y;
  const int thread_idx = threadIdx.x;

  const int head_offset = z * gridDim.y * dim + y * dim;

  extern __shared__ float data[];
  for (int i = 0; i < N; ++i) {
    int offset = thread_idx * N + i;
    if (offset < dim && head_offset + offset < dsize) {
      data[offset] = x1_data[head_offset + offset];
    } else {
      data[offset] = 0;
    }
  }
  __syncthreads();
  for (int gap = (blockDim.x * N) >> 1; gap > 0; gap >>= 1) {
    for (int i = 0; i < N; ++i) {
      int offset = thread_idx * N + i;
      if (offset < gap) {
        data[offset] += data[offset + gap];
      }
    }
    __syncthreads();
  }
  __syncthreads();
  float mean = data[0] / dim;

  for (int i = 0; i < N; ++i) {
    int offset = thread_idx * N + i;
    if (offset < dim && head_offset + offset < dsize) {
      data[offset] = (x1_data[head_offset + offset] - mean) *
                     (x1_data[head_offset + offset] - mean);
    } else {
      data[offset] = 0;
    }
  }
  __syncthreads();
  for (int gap = blockDim.x * N >> 1; gap > 0; gap >>= 1) {
    for (int i = 0; i < N; ++i) {
      int offset = thread_idx * N + i;
      if (offset < gap) {
        data[offset] += data[offset + gap];
      }
    }
    __syncthreads();
  }
  __syncthreads();
  float accum = data[0];
  float stdev = sqrt(accum / dim + ep);

  float a1 = -(1 / (dim * stdev));

  for (int i = 0; i < N; ++i) {
    int offset = thread_idx * N + i;
    if (offset < dim && head_offset + offset < dsize) {
      data[offset] = x_grad[head_offset + offset];
    } else {
      data[offset] = 0;
    }
  }
  __syncthreads();
  for (int gap = blockDim.x * N >> 1; gap > 0; gap >>= 1) {
    for (int i = 0; i < N; ++i) {
      int offset = thread_idx * N + i;
      if (offset < gap) {
        data[offset] += data[offset + gap];
      }
    }
    __syncthreads();
  }
  __syncthreads();
  float grad_accum = data[0] * a1;

  for (int i = 0; i < N; ++i) {
    int offset = thread_idx * N + i;
    if (offset < dim && head_offset + offset < dsize) {
      data[offset] =
          x_grad[head_offset + offset] * x_data[head_offset + offset];
    } else {
      data[offset] = 0;
    }
  }
  __syncthreads();
  for (int gap = blockDim.x * N >> 1; gap > 0; gap >>= 1) {
    for (int i = 0; i < N; ++i) {
      int offset = thread_idx * N + i;
      if (offset < gap) {
        data[offset] += data[offset + gap];
      }
    }
    __syncthreads();
  }
  __syncthreads();
  float grad_data_accum = data[0] * a1;

  // for (int i = 0; i < N; ++i) {
  //     int offset = thread_idx * N + i;
  //     if (offset < dim && head_offset + offset < dsize) {
  //         data[offset] = x1_data[head_offset + offset];
  //     }
  //     else {
  //         data[offset] = 0;
  //     }
  // }
  // __syncthreads();

  for (int i = 0; i < N; ++i) {
    int offset = thread_idx * N + i;
    if (offset < dim && head_offset + offset < dsize) {
      x1_grad[head_offset + offset] += x_grad[head_offset + offset] / stdev;
      x1_grad[head_offset + offset] +=
          grad_accum + grad_data_accum * x_data[head_offset + offset];
      // if (std::isnan(x1_grad[head_offset + offset])) {
      //   printf("%.2f %.2f %.2f %.2f %.2f\n", stdev, p1, p2,
      //   x1_data[head_offset + offset], mean);
      // }
    }
  }
}

void Dropout::forward_process(Tensor& x1, Tensor& x) {
#ifdef EVAL
  x = x1 * 1;
  return;
#endif
  if (x1.device() == "cpu") {
    for (int i = 0; i < x1.dsize(); ++i) {
      if (di(dre) < limit_) {
        x[i] = 0;
      } else {
        x[i] = x1[i] / (1 - prob_);
      }
    }
  } else {
    const int block_size = 512;
    const int grid_size = (x1.dsize() + 511) / 512;
    drouput_kernel<<<grid_size, block_size>>>(x1.data(), x.data(), prob_,
                                              x1.dsize(), time(nullptr));
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

__global__ void drouput_kernel(float* x1, float* x, float prob, index_t dsize,
                               unsigned long long seed) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < dsize) {
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    if (hiprand_uniform(&state) < prob) {
      x[idx] = 0;
    } else {
      x[idx] = x1[idx] / (1 - prob);
    }
  }
}

void Dropout::grad_fn(TensorImplPtr x1, TensorImplPtr x) {
  if (x1->device() == "cpu") {
#ifdef EVAL
    memcpy(x1->grad_, x->grad_, sizeof(float) * x->dsize());
    return;
#endif
    for (int i = 0; i < x1->dsize(); ++i) {
      if (fabs((*x)[i]) < 1e-6) {
        continue;
      } else {
        x1->grad_[i] += (x->grad_[i] / (1 - prob_));
      }
    }
  } else {
#ifdef EVAL
    hipMemcpy(x1->grad_, x->grad_, sizeof(float) * x->dsize(),
               hipMemcpyDeviceToDevice);
    return;
#endif
    const int block_size = 512;
    const int grid_size = (x1->dsize() + 511) / 512;
    dropout_backward_kernel<<<grid_size, block_size>>>(
        x1->grad_, x->grad_, x->data_, prob_, x1->dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

__global__ void dropout_backward_kernel(float* x1_grad, float* x_grad,
                                        float* x_data, float prob,
                                        index_t dsize) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < dsize) {
    if (fabs(x_data[idx]) < 1e-6) {
      return;
    } else {
      x1_grad[idx] += x_grad[idx] / (1 - prob);
    }
  }
}

void GELU::forward_process(Tensor& x1, Tensor& x) {
  if (x1.device() == "cpu") {
    const float n = sqrt(2 / M_PI);
    for (int i = 0; i < x1.dsize(); ++i) {
      x[i] = 0.5 * x1[i] * (1 + tanh(n * (x1[i] + 0.044715 * pow(x1[i], 3))));
    }
  } else {
    const int block_size = 512;
    const int grid_size = (x1.dsize() + 511) / 512;
    gelu_kernel<<<grid_size, block_size>>>(x1.data(), x.data(), x1.dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

__global__ void gelu_kernel(float* x1, float* x, index_t dsize) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < dsize) {
    const float n = sqrt(2 / M_PI);
    x[idx] =
        0.5 * x1[idx] * (1 + tanh(n * (x1[idx] + 0.044715 * pow(x1[idx], 3))));
  }
}

void GELU::grad_fn(TensorImplPtr x1, TensorImplPtr x) {
  if (x1->device() == "cpu") {
    const float n = sqrt(2 / M_PI);
    for (int i = 0; i < x1->dsize(); ++i) {
      const float th = tanh(n * ((*x1)[i] + 0.044715 * pow((*x1)[i], 3)));
      x1->grad_[i] +=
          x->grad_[i] *
          (0.5 * (1 + th) + 0.5 * (*x1)[i] * (1 - th * th) * n *
                                (1 + 0.044715 * 3 * (*x1)[i] * (*x1)[i]));
    }
  } else {
    const int block_size = 512;
    const int grid_size = (x1->dsize() + 511) / 512;
    gelu_backward_kernel<<<grid_size, block_size>>>(x1->grad_, x1->data_,
                                                    x->grad_, x1->dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

__global__ void gelu_backward_kernel(float* x1_grad, float* x1_data,
                                     float* x_grad, index_t dsize) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < dsize) {
    const float n = sqrt(2 / M_PI);
    const float th = tanh(n * (x1_data[idx] + 0.044715 * pow(x1_data[idx], 3)));
    x1_grad[idx] +=
        x_grad[idx] *
        (0.5 * (1 + th) + 0.5 * x1_data[idx] * (1 - th * th) * n *
                              (1 + 0.044715 * 3 * x1_data[idx] * x1_data[idx]));
  }
}

__global__ void before_softmax_kernel(float* x1, index_t dim,
                                      index_t sum_size) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < sum_size) {
    index_t offset = idx * dim;
    float max_num = x1[offset];
    for (int i = 0; i < dim; ++i) {
      max_num = max(max_num, x1[offset + i]);
    }
    for (int i = 0; i < dim; ++i) {
      x1[offset + i] -= max_num;
    }
  }
}

void Softmax::before_softmax(Tensor& x1) {
  if (x1.device() == "cpu") {
    index_t dim = x1.shape()[x1.shape().size() - 1];
    for (int b = 0; b < x1.dsize() / dim; ++b) {
      index_t offset = b * dim;
      float max_num = x1[offset];
      for (int i = 0; i < dim; ++i) {
        max_num = max(max_num, x1[offset + i]);
      }
      for (int i = 0; i < dim; ++i) {
        x1[offset + i] -= max_num;
      }
    }
  } else {
    index_t dim = x1.shape()[x1.shape().size() - 1];
    const int block_size = 512;
    const int grid_size = (x1.dsize() / dim + 511) / 512;
    before_softmax_kernel<<<grid_size, block_size>>>(x1.data(), dim,
                                                     x1.dsize() / dim);
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

void Softmax::forward_process(Tensor& x1, Tensor& x) {
  // before_softmax(x1);
  if (x1.device() == "cpu") {
    index_t dim = x1.shape()[x1.shape().size() - 1];
    for (int b = 0; b < x1.dsize() / dim; ++b) {
      index_t offset = b * dim;
      float exp_sum = 0;
      for (int i = 0; i < dim; ++i) {
        exp_sum += exp(x1[offset + i]);
      }
      for (int i = 0; i < dim; ++i) {
        x[offset + i] = exp(x1[offset + i]) / exp_sum;
      }
    }
  } else {
    auto shape = x.shape();
    // hipMemcpy(x.data(), x1.data(), sizeof(float) * x1.dsize(),
    // hipMemcpyDeviceToDevice);
    const dim3 grid_size(
        1, shape[shape.size() - 2],
        x.dsize() / (shape[shape.size() - 2] * shape[shape.size() - 1]));
    const int block_size = min(
        512, (shape[shape.size() - 1] % 2 == 0 ? shape[shape.size() - 1]
                                               : shape[shape.size() - 1] + 1));
    softmax_kernel<<<grid_size, block_size,
                     sizeof(float) *
                         ((shape[shape.size() - 1] + block_size - 1) /
                          block_size * block_size)>>>(
        x1.data(), x.data(),
        (shape[shape.size() - 1] + block_size - 1) / block_size,
        shape[shape.size() - 1], x.dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

__global__ void softmax_kernel(float* x1, float* x, int N, index_t dim,
                               index_t dsize) {
  const index_t z = blockIdx.z;
  const index_t y = blockIdx.y;
  const index_t thread_idx = threadIdx.x;

  const index_t head_offset = z * gridDim.y * dim + y * dim;

  extern __shared__ float data[];
  for (int i = 0; i < N; ++i) {
    index_t offset = thread_idx * N + i;
    if (offset < dim && (head_offset + offset) < dsize) {
        data[offset] = exp(x1[head_offset + offset]);
    } else {
      data[offset] = 0;
    }
  }
  __syncthreads();
  for (int gap = (blockDim.x * N) >> 1; gap > 0; gap >>= 1) {
    for (int i = 0; i < N; ++i) {
      int offset = thread_idx * N + i;
      if (offset < gap) {
        data[offset] += data[offset + gap];
      }
    }
    __syncthreads();
  }
  __syncthreads();
  float exp_sum = data[0];

  for (int i = 0; i < N; ++i) {
    int offset = thread_idx * N + i;
    if (offset < dim && head_offset + offset < dsize) {
      x[head_offset + offset] = exp(x1[head_offset + offset]) / exp_sum;
      assert(x[head_offset + offset] < 1);
    }
  }
}

void Softmax::grad_fn(TensorImplPtr x1, TensorImplPtr x) {
  if (x1->device() == "cpu") {
    index_t dim = x1->shape()[x1->shape().size() - 1];
    for (int b = 0; b < x1->dsize() / dim; ++b) {
      index_t offset = b * dim;
      float sum = 0;
      for (int i = 0; i < dim; ++i) {
        sum += x->data_[offset + i] * x->grad_[offset + i];
      }
      for (int i = 0; i < dim; ++i) {
        x1->grad_[offset + i] += (x->data_[offset + i] * x->grad_[offset + i] -
                                  x->data_[offset + i] * sum);
      }
    }
  } else {
    auto shape = x->shape();
    const dim3 grid_size(
        1, shape[shape.size() - 2],
        x->dsize() / (shape[shape.size() - 2] * shape[shape.size() - 1]));
    const int block_size = min(
        512, (shape[shape.size() - 1] % 2 == 0 ? shape[shape.size() - 1]
                                               : shape[shape.size() - 1] + 1));
    softmax_backward_kernel<<<grid_size, block_size,
                              3 * sizeof(float) *
                                  ((shape[shape.size() - 1] + block_size - 1) /
                                   block_size * block_size)>>>(
        x1->grad_, x1->data_, x->grad_, x->data_,
        (shape[shape.size() - 1] + block_size - 1) / block_size,
        shape[shape.size() - 1], x->dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

__global__ void softmax_backward_kernel(float* x1_grad, float* x1_data,
                                        float* x_grad, float* x_data, const int N,
                                        index_t dim, index_t dsize) {
  const index_t z = blockIdx.z;
  const index_t y = blockIdx.y;
  const index_t thread_idx = threadIdx.x;

  const index_t head_offset = z * gridDim.y * dim + y * dim;

  extern __shared__ float shared_data[];
  float* x_data_s = shared_data;
  float* x_grad_s = (float*)&x_data_s[N * blockDim.x];
  float* data = (float*)&x_grad_s[N * blockDim.x];

  for (int i = 0; i < N; ++i) {
    index_t offset = thread_idx * N + i;
    if (offset < dim && (head_offset + offset) < dsize) {
        x_data_s[offset] = x_data[head_offset + offset];
        x_grad_s[offset] = x_grad[head_offset + offset];
    } else {
      x_data_s[offset] = 0;
      x_grad_s[offset] = 0;
    }
  }
  __syncthreads();

  for (int i = 0; i < N; ++i) {
    index_t offset = thread_idx * N + i;
    if (offset < dim && (head_offset + offset) < dsize) {
        data[offset] =
            x_data_s[offset] * x_grad_s[offset];
    } else {
      data[offset] = 0;
    }
  }
  __syncthreads();
  for (int gap = (blockDim.x * N) >> 1; gap > 0; gap >>= 1) {
    for (int i = 0; i < N; ++i) {
      int offset = thread_idx * N + i;
      if (offset < gap) {
        data[offset] += data[offset + gap];
      }
    }
    __syncthreads();
  }
  __syncthreads();
  float sum = data[0];

  for (int i = 0; i < N; ++i) {
    int offset = thread_idx * N + i;
    if (offset < dim && head_offset + offset < dsize) {
      x1_grad[head_offset + offset] +=
          (x_data_s[offset] * x_grad_s[offset] -
           x_data_s[offset] * sum);
    }
  }
}

void Log::forward_process(Tensor& x1, Tensor& x) {
  if (x1.device() == "cpu") {
    for (int i = 0; i < x1.dsize(); ++i) {
      x[i] = log(x1[i]);
    }
  } else {
    const int block_size = 512;
    const int grid_size = (x1.dsize() + 511) / 512;
    log_kernel<<<grid_size, block_size>>>(x1.data(), x.data(), x1.dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

__global__ void log_kernel(float* x1, float* x, index_t dsize) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < dsize) {
    x[idx] = log(x1[idx]);
  }
}

void Log::grad_fn(TensorImplPtr x1, TensorImplPtr x) {
  if (x1->device() == "cpu") {
    for (int i = 0; i < x1->dsize(); ++i) {
      x1->grad_[i] += (x->grad_[i] / (x1->data_[i] + 1e-8));
    }
  } else {
    const int block_size = 512;
    const int grid_size = (x1->dsize() + 511) / 512;
    log_backward_kernel<<<grid_size, block_size>>>(x1->grad_, x1->data_,
                                                   x->grad_, x1->dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

__global__ void log_backward_kernel(float* x1_grad, float* x1_data,
                                    float* x_grad, index_t dsize) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < dsize) {
    x1_grad[idx] += (x_grad[idx] / (x1_data[idx] + 1e-8));
  }
}

Tensor NLLLoss::generate_ret_tensor(Tensor& x1, Tensor& x2) {
  Tensor ret({1});
  ret.data()[0] = 0;
  ret.grad()[0] = 1;
  ret.to(x1.device());
  return ret;
}

void NLLLoss::prepare_forward(Tensor& x1, Tensor& x2, Tensor& x) {
  // 1. check
  if (x1.device() != x2.device()) {
    std::cout << "TensorImplPtr must in same device." << std::endl;
    assert(false);
  }

  assert(x1.ndim() == 2);
  assert(x2.ndim() == 1);
  assert(x1.shape()[0] == x2.dsize());

  // 2. add node to return Tensor
  GraphNodePtr node = std::make_shared<BinaryGraphNode>(x1.get(), x2.get());
  node->setGradFnL(std::bind(&NLLLoss::lhs_grad_fn, this, _1, _2, _3));
  node->setGradFnR(std::bind(&NLLLoss::rhs_grad_fn, this, _1, _2, _3));
  x.setNode(node);

  return;
}

void NLLLoss::forward_process(Tensor& x1, Tensor& x2, Tensor& x) {
  if (x1.device() == "cpu") {
    for (int i = 0; i < x2.dsize(); ++i) {
      index_t offset = x1.shape()[1] * i;
      x[0] += (-x1[offset + (int)x2[i]]);
    }
    x[0] /= x2.dsize();
  } else {
    const int block_size = 512;
    const int grid_size = (x2.dsize() + 511) / 512;
    nllloss_kernel<<<grid_size, block_size, sizeof(float) * 512>>>(
        x1.data(), x2.data(), x.data(), x1.shape()[1], x2.dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
    x.cpu();
    hipDeviceSynchronize();
    x[0] /= x2.dsize();
    x.cuda();
  }
}

__global__ void nllloss_kernel(float* x1, float* x2, float* x, index_t len,
                               index_t dsize) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  extern __shared__ float data[];
  if (idx < dsize) {
    data[threadIdx.x] = -x1[idx * len + (int)x2[idx]];
  } else {
    data[threadIdx.x] = 0;
  }
  __syncthreads();

  for (int gap = blockDim.x >> 1; gap > 0; gap >>= 1) {
    if (threadIdx.x < gap) {
      data[threadIdx.x] += data[threadIdx.x + gap];
    }
    __syncthreads();
  }
  __syncthreads();
  float loss = data[0];

  if (threadIdx.x == 0) {
    atomicAdd(x, loss);
  }
  __syncthreads();
  // if (idx == 0) {
  //     x[0] /= dsize;
  // }
  // __syncthreads();
}

void NLLLoss::lhs_grad_fn(TensorImplPtr x1, TensorImplPtr x2, TensorImplPtr x) {
  if (x1->device() == "cpu") {
    for (int i = 0; i < x2->dsize(); ++i) {
      index_t offset = x1->shape()[1] * i;
      x1->grad_[offset + (int)(x2->data_[i])] += -(
          x->grad_[0] / x2->dsize() * x1->data_[offset + (int)(x2->data_[i])]);
    }
  } else {
    const int block_size = 512;
    const int grid_size = (x2->dsize() + 511) / 512;
    nllloss_backward_kernel<<<grid_size, block_size>>>(
        x1->grad_, x1->data_, x2->data_, x->grad_, x1->shape()[1], x2->dsize());
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}

void NLLLoss::rhs_grad_fn(TensorImplPtr x1, TensorImplPtr x2, TensorImplPtr x) {
  return;
}

__global__ void nllloss_backward_kernel(float* x1_grad, float* x1_data,
                                        float* x2_data, float* x_grad,
                                        index_t len, index_t dsize) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < dsize) {
    x1_grad[idx * len + (int)(x2_data[idx])] +=
        -(x_grad[0] / dsize * x1_data[idx * len + (int)(x2_data[idx])]);
  }
}

Tensor Emb::generate_ret_tensor(Tensor& idx, Tensor& emb) {
  if (idx.shape().size() == 1) {
    Tensor x({idx.shape()[0], emb.shape()[1]}, emb.device());
    return x;
  } else {
    Tensor x({idx.shape()[0], idx.shape()[1], emb.shape()[1]}, emb.device());
    return x;
  }
}

void Emb::prepare_forward(Tensor& idx, Tensor& emb, Tensor& x) {
  // 1. check
  if (idx.device() != emb.device()) {
    std::cout << "TensorImplPtr must in same device." << std::endl;
    assert(false);
  }

  assert(emb.shape().size() == 2);

  // 2. add node to return Tensor
  GraphNodePtr node = std::make_shared<BinaryGraphNode>(idx.get(), emb.get());
  node->setGradFnL(std::bind(&Emb::lhs_grad_fn, this, _1, _2, _3));
  node->setGradFnR(std::bind(&Emb::rhs_grad_fn, this, _1, _2, _3));
  x.setNode(node);
}

void Emb::forward_process(Tensor& idx, Tensor& emb, Tensor& x) {
  index_t vocab_size = emb.shape()[0];
  index_t hidden_dim = emb.shape()[1];
  if (idx.device() == "cpu") {
    for (int i = 0; i < idx.dsize(); ++i) {
      index_t offset = (index_t)idx[i];
      assert(offset < vocab_size);
      memcpy(x.data() + hidden_dim * i, emb.data() + offset * hidden_dim,
             sizeof(float) * hidden_dim);
    }
  } else {
    idx.cpu();
    for (int i = 0; i < idx.dsize(); ++i) {
      index_t offset = (index_t)idx[i];
      assert(offset < vocab_size);
      hipMemcpy(x.data() + hidden_dim * i, emb.data() + offset * hidden_dim,
                 sizeof(float) * hidden_dim, hipMemcpyDeviceToDevice);
    }
    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
    idx.cuda();
  }
}

void Emb::lhs_grad_fn(TensorImplPtr x1, TensorImplPtr x2, TensorImplPtr x) {
  return;
}

__global__ void emb_backward_kernel(float* idx, float* emb_grad, float* x_grad,
                                    int N, index_t dim, index_t dsize) {
  const index_t id = blockIdx.x;
  const index_t thread_idx = threadIdx.x;

  if (id < dsize) {
    index_t vocab_id = (index_t)idx[id];
    index_t head_offset = thread_idx * N;
    for (int i = 0; i < N; ++i) {
      if (head_offset + i < dim) {
        atomicAdd(emb_grad + vocab_id * dim + head_offset + i,
                  x_grad[id * dim + head_offset + i]);
      }
    }
  }
  __syncthreads();
}

void Emb::rhs_grad_fn(TensorImplPtr idx, TensorImplPtr emb, TensorImplPtr x) {
  index_t hidden_dim = emb->shape()[1];
  if (idx->device() == "cpu") {
    for (int i = 0; i < idx->dsize(); ++i) {
      index_t offset = (index_t)idx->data_[i];
      for (int d = 0; d < hidden_dim; ++d) {
        emb->grad_[offset * hidden_dim + d] += x->grad_[i * hidden_dim + d];
      }
    }
  } else {
    int grid_size = idx->dsize();
    int block_size = min(512, (emb->shape()[1] % 2 == 0 ? emb->shape()[1]
                                                        : emb->shape()[1] + 1));
    emb_backward_kernel<<<grid_size, block_size>>>(
        idx->data_, emb->grad_, x->grad_,
        (hidden_dim + block_size - 1) / block_size, hidden_dim, idx->dsize());

    hipDeviceSynchronize();
    auto error = hipGetLastError();
    if (hipSuccess != error) {
      printf("%s\n", hipGetErrorString(error));
      assert(false);
    }
  }
}